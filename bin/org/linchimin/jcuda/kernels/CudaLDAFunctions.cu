

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>


/**
 * Some kernel functions for LDA
 * 
 * @author Lin Chi-Min (v381654729@gmail.com)
 */


/**
 * wpt[k][v] : words per topic
 * wt[k] : total words per topic
 * tpd[m][k] : topics per document
 * td[m] : total topics per document
 * 
 * wpt and phis : a K * V matrix
 * tpd and thetas : a M * K matrix
 * 
 * numElements = (num documents in one batch) 
 * 
 * p:  K * numDocumentsInOneBatch
 */
extern "C"
__global__ void drawLatentVariables(
		const int* __restrict__ docsWordCounts, const int* __restrict__ docsWordOffsets, const int* __restrict__ docsWordIndices,
		int* wpt, int* wt, int* tpd, int* td, const float* __restrict__ phis, const float* __restrict__ thetas, 
		float* p,
		int docOffset, int K, int M, int V, int numDocumentsInOneBatch) {
	
	/**
	 * with size numDocumentsInOneBatch * K
	 */
	
	int m = blockDim.x * blockIdx.x + threadIdx.x;
    if (m < numDocumentsInOneBatch) {
    	
    	int Nm = docsWordCounts[m];
    	int docIndex = docOffset + m;
    	int docWordOffset = docsWordOffsets[m];
    	
    	hiprandState s;
    	// reset a random number generator
    	hiprand_init(docIndex, 0, 0, &s);
    	
    	int pOffset = m * K;
    	
    	for (int i = 0; i < Nm; i++) {
    		
    		float sum = 0;
    		int c_word = docsWordIndices[docWordOffset + i];
    		if (c_word < 0 || c_word >= V){
    			continue;
    		}
    		int j;
    		for (j = 0; j < K; j++) {
    			sum += phis[j + (c_word * K)] * thetas[docIndex + (j * M)];
    			p[j + pOffset] = sum;
			}
    		float stop = hiprand_uniform(&s) * sum;
    		for (j = 0; j < K; j++) {
    			if (stop < p[j + pOffset]) {
    				break;
    			}
			}
    		if (j == K){
    			j--;
    		}
    		
    		atomicAdd(&wpt[j + (c_word * K)], 1);
    		atomicAdd(&wt[j], 1);
    		tpd[docIndex + (j * M)]++;
		}
    	td[docIndex] += Nm;
    }
	
}


extern "C"
__global__ void drawLatentVariablesForTestingQuick(
		const int* __restrict__ docsWordCounts, const int* __restrict__ docsWordOffsets, const int* __restrict__ docsWordIndices,
		int* tpd, int* td, const float* __restrict__ phis, const float* __restrict__ thetas, 
		int docOffset, int K, int M, int numDocumentsInOneBatch) {
	
	extern __shared__ float p[];
	
	/**
	 * with size numDocumentsInOneBatch * K
	 */
	
	int m = blockDim.x * blockIdx.x + threadIdx.x;
    if (m < numDocumentsInOneBatch) {
    	
    	int Nm = docsWordCounts[m];
    	int docIndex = docOffset + m;
    	int docWordOffset = docsWordOffsets[m];
    	
    	hiprandState s;
    	// reset a random number generator
    	hiprand_init(docIndex, 0, 0, &s);
    	
    	int pOffset = m * K;
    	
    	for (int i = 0; i < Nm; i++) {
    		
    		float sum = 0;
    		int c_word = docsWordIndices[docWordOffset + i];
    		if (c_word < 0){
    			continue;
    		}
    		int j;
    		for (j = 0; j < K; j++) {
    			sum += phis[j + (c_word * K)] * thetas[docIndex + (j * M)];
    			p[j + pOffset] = sum;
			}
    		float stop = hiprand_uniform(&s) * sum;
    		for (j = 0; j < K; j++) {
    			if (stop < p[j + pOffset]) {
    				break;
    			}
			}
    		if (j == K){
    			j--;
    		}
    		
    		tpd[docIndex + (j * M)]++;
		}
    	td[docIndex] += Nm;
    }
	
}



/**
 * Use this to infer topics for testing;
 * phis are fixed and not updated 
 * 
 */
extern "C"
__global__ void drawLatentVariablesForTesting(
		const int* __restrict__ docsWordCounts, const int* __restrict__ docsWordOffsets, const int* __restrict__ docsWordIndices,
		int* tpd, int* td, const float* __restrict__ phis, const float* __restrict__ thetas, 
		float* p,
		int docOffset, int K, int M, int numDocumentsInOneBatch) {
	
	/**
	 * with size numDocumentsInOneBatch * K
	 */
	
	int m = blockDim.x * blockIdx.x + threadIdx.x;
    if (m < numDocumentsInOneBatch) {
    	
    	int Nm = docsWordCounts[m];
    	int docIndex = docOffset + m;
    	int docWordOffset = docsWordOffsets[m];
    	
    	hiprandState s;
    	// reset a random number generator
    	hiprand_init(docIndex, 0, 0, &s);
    	
    	int pOffset = m * K;
    	
    	for (int i = 0; i < Nm; i++) {
    		
    		float sum = 0;
    		int c_word = docsWordIndices[docWordOffset + i];
    		if (c_word < 0){
    			continue;
    		}
    		int j;
    		for (j = 0; j < K; j++) {
    			sum += phis[j + (c_word * K)] * thetas[docIndex + (j * M)];
    			p[j + pOffset] = sum;
			}
    		float stop = hiprand_uniform(&s) * sum;
    		for (j = 0; j < K; j++) {
    			if (stop < p[j + pOffset]) {
    				break;
    			}
			}
    		if (j == K){
    			j--;
    		}
    		
    		tpd[docIndex + (j * M)]++;
		}
    	td[docIndex] += Nm;
    }
	
}



/**
 * K = 30
 * V = 120000
 * 
 * numElements = K * V
 * 
 * wpt and phis : a K * V matrix
 * 
 * wpt[k][v] : words per topic
 * wt[k] : total words per topic
 * 
 */
extern "C"
__global__ void computePhis(const int* __restrict__ wpt, const int* __restrict__ wt, float* phis, float beta, float betaV, int K, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < numElements) {	
		int k = i % K;
		phis[i] = (wpt[i] + beta) / (wt[k] + betaV);
	}
}


extern "C"
__global__ void computePhisExact(const int* __restrict__ wpt, const int* __restrict__ wt, float* phis, int K, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {	
    	int k = i % K;
    	phis[i] = (wpt[i] + 0.0) / wt[k];
    }
}



/**
 * M = 90000
 * K = 30
 * 
 * numElements = M * K
 * 
 * thetas : a M * K matrix
 * 
 * tpd[m][k] : topics per document
 * td[m] : total topics per document
 */
extern "C"
__global__ void computeThetas(const int* __restrict__  tpd, const int* __restrict__ td, float* thetas, 
		float alpha, float alphaK, int M, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
    	int m = i % M;
    	thetas[i] = (tpd[i] + alpha) / (td[m] + alphaK);
    }
}





